#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>

#include <device_types.h>
#include <mma.h>
#include <hip/hip_vector_types.h>

#include <mlkl/core/basic_math.h>
#include <mlkl/operators/cuda/bf16_gemm.h>

// TODO: Delete this and make functions templates
#define WARP_SIZE 32

using namespace nvcuda;

namespace mlkl::operators::cuda {
namespace kernel {
// naive
template<int WM, int WN, int WK>
__global__ __launch_bounds__(32) void attention_v1(const fp32 *q, const fp32 *k, fp32 *v, size_t M, size_t N, size_t K) {
  // suppose we are processing block (2, 3) and WM = WN = WK = 16
  // warp_col = 2 * WN = 2 * 16 = 32
  // warp row = 3 * WN = 3 * 16 = 48
  //
  // block (7, 7)
  // warp_col = 7 * WN = 7 * 16 = 112
  // warp row = 7 * WN = 7 * 16 = 112
  int warp_row = blockIdx.y * WM;
  int warp_col = blockIdx.x * WN;

  wmma::fragment<wmma::matrix_a, WM, WN, WK, bf16, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, WM, WN, WK, bf16, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, WM, WN, WK, float> accumulator;
  wmma::fill_fragment(accumulator, 0.);

  for (int step = 0; step < K / WK; ++step) {

    wmma::load_matrix_sync(a_frag, &a[(warp_row * WM) * K + (step * WK)], K);
    wmma::load_matrix_sync(b_frag, &b[(step * WK) * N + (warp_col * WN)], N);

    wmma::mma_sync(accumulator, a_frag, b_frag, accumulator);
  }

  wmma::store_matrix_sync(&c[warp_row * N + warp_col], accumulator, N, wmma::mem_row_major);
}
}// namespace kernel

namespace {
void launch_attention_v1(const fp32 *a, float alpha, const fp32 *b, float beta, fp32 *c, size_t M, size_t N, size_t K) {
  constexpr int WM = 16;
  constexpr int WN = 16;
  constexpr int WK = 16;

  // if M = 128, M = 128
  // (128 / WN, 128 / WM) = (128 / 16, 128 / 16)
  // so we will launch a 8x8 grid where each block is comprised
  // a collection of 32-threads
  dim3 block_dim(WARP_SIZE);
  dim3 grid_dim(math::ceil_div(N, WN), math::ceil_div(M, WM));

  kernel::attention_v1<WM, WN, WK><<<grid_dim, block_dim>>>(a, alpha, b, beta, c, M, N, K);
}
}// namespace

void attention_v1(Tensor *q, Tensor *k, Tensor *v, Tensor *output) {
  launch_attention_v1(q->fp32_(), k->fp32_(), v->fp32_(), c->shape[0], c->shape[1], a->shape[1]);
}

void attention(Tensor *q, Tensor *k, Tensor *v, Tensor *output) {
  attention_v1(q, k, v, output);
}
}// namespace mlkl::operators::cuda
