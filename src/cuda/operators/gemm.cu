#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void gemm(const float *a, float alpha, const float *b, float beta, float *c, size_t M, size_t N, size_t K) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.x;

  for (int)
}