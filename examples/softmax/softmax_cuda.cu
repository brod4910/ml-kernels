#include "mlkl/core/tensor.h"
#include <mlkl/mlkl.h>

#include <cassert>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

template<typename Kernel>
void test_kernel(const char *kernel_name,
                 Kernel kernel,
                 int M, int N, int num_runs = 10) {
  std::vector<int>
    shape{M, N};
  auto cpu_allocator = mlkl::TensorAllocator(mlkl::Device::CPU);
  auto gpu_allocator = mlkl::TensorAllocator(mlkl::Device::CUDA);

  std::vector<int> s1{M, N};

  auto a_d = gpu_allocator.randn(s1);
  auto b_d = gpu_allocator.randn(s1);

  auto a_cpu = cpu_allocator.empty(s1);
  auto b_cpu = cpu_allocator.empty(s1);

  mlkl::softmax(a_cpu, b_cpu, 0, mlkl::Device::CPU);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float total_duration = 0;

  // warm-up
  for (int i = 0; i < 10; ++i) {
    kernel(a_d, b_d, 0, shape);
    CHECK_CUDA_ERROR();
  }

  for (int i = 0; i < num_runs; ++i) {
    hipMemcpy(b_d, b, M * N * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();

    hipEventRecord(start);

    kernel(a_d, b_d, 0, shape);
    CHECK_CUDA_ERROR();

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_elapsed;
    hipEventElapsedTime(&time_elapsed, start, stop);
    total_duration += time_elapsed;
  }

  hipMemcpy(b, b_d, M * N * sizeof(float), hipMemcpyDeviceToHost);
  CHECK_CUDA_ERROR();

  bool correct = mlkl::cpu::utils::assert_correctness(b, ref_matrix, M, N);
  if (!correct) {
    std::cerr << "Kernel " << kernel_name << " produced incorrect results." << std::endl;
  }

  float average_duration = total_duration / num_runs;
  float gflops = (2.0f * M * N) / (average_duration / 1000.0f) / 1e9;

  std::cout << "Kernel: " << kernel_name << " | "
            << "Size: " << M << "x" << N << " | "
            << "Time: " << average_duration << " ms | "
            << "GFLOPS: " << gflops << std::endl;

  // std::cout << "matrix: \n";
  // print_matrix(c, M, N);
  // std::cout << "ref: \n";
  // print_matrix(ref_matrix, M, N);

  // Cleanup
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(a_d);
  hipFree(b_d);
  delete[] a;
  delete[] b;
  delete[] ref_matrix;
}

void softmax_cuda(int M, int N) {
  int num_runs = 1000;

  // Test custom kernels
  test_kernel("Softmax Kernel V1", [&](float *a, float *b, int dim, std::vector<int> &shape) { mlkl::operators::cuda::launch_softmax_2d_v1(a, b, dim, shape); }, M, N, num_runs);
}