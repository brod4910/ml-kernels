#include "mlkl/tensor/create.h"
#include "mlkl/tensor/tensor.h"
#include <mlkl/mlkl.h>

#include <cassert>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

template<typename Kernel>
void test_kernel(const char *kernel_name,
                 Kernel kernel,
                 int M, int N, int num_runs = 10) {
  auto a = mlkl::create_tensor<float>(shape.data(), mlkl::CPU);
  auto b = mlkl::create_tensor<float>(shape.data(), mlkl::CPU);
  auto *ref_matrix = new float[M * N];

  std::vector<int>
    shape{M, N};
  mlkl::cpu::utils::set_random_matrix(a, M, N);
  mlkl::cpu::utils::set_random_matrix(b, M, N);

  mlkl::cpu::utils::fill_matrix(b, M, N, 0);
  mlkl::cpu::utils::fill_matrix(ref_matrix, M, N, 0);
  ml::operators::cpu::softmax(a, b, 0, shape);

  auto *a_d = initialize_cuda_matrix(a, M * M);
  auto *b_d = initialize_cuda_matrix(b, M * N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float total_duration = 0;

  // warm-up
  for (int i = 0; i < 10; ++i) {
    kernel(a_d, b_d, 0, shape);
    CHECK_CUDA_ERROR();
  }

  for (int i = 0; i < num_runs; ++i) {
    hipMemcpy(b_d, b, M * N * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();

    hipEventRecord(start);

    kernel(a_d, b_d, 0, shape);
    CHECK_CUDA_ERROR();

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_elapsed;
    hipEventElapsedTime(&time_elapsed, start, stop);
    total_duration += time_elapsed;
  }

  hipMemcpy(b, b_d, M * N * sizeof(float), hipMemcpyDeviceToHost);
  CHECK_CUDA_ERROR();

  bool correct = mlkl::cpu::utils::assert_correctness(b, ref_matrix, M, N);
  if (!correct) {
    std::cerr << "Kernel " << kernel_name << " produced incorrect results." << std::endl;
  }

  float average_duration = total_duration / num_runs;
  float gflops = (2.0f * M * N) / (average_duration / 1000.0f) / 1e9;

  std::cout << "Kernel: " << kernel_name << " | "
            << "Size: " << M << "x" << N << " | "
            << "Time: " << average_duration << " ms | "
            << "GFLOPS: " << gflops << std::endl;

  // std::cout << "matrix: \n";
  // print_matrix(c, M, N);
  // std::cout << "ref: \n";
  // print_matrix(ref_matrix, M, N);

  // Cleanup
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(a_d);
  hipFree(b_d);
  delete[] a;
  delete[] b;
  delete[] ref_matrix;
}

void softmax_cuda(int M, int N) {
  int num_runs = 1000;

  // Test custom kernels
  test_kernel("Softmax Kernel V1", [&](float *a, float *b, int dim, std::vector<int> &shape) { ml::operators::cuda::launch_softmax_2d_v1(a, b, dim, shape); }, M, N, num_runs);
}