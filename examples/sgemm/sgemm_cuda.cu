#include "mlkl/core/tensor_ops.h"
#include <mlkl/mlkl.h>
#include <mlkl/operators/cuda/gemm.h>
#include <mlkl/utils/device.h>

#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>

#define CHECK_CUBLAS_STATUS(val) checkCuBLASStatus((val), #val, __FILE__, __LINE__)
void checkCuBLASStatus(hipblasStatus_t status, const char *const func, const char *const file, const int line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS Error at : " << file << ":" << line << '\n';
    std::cerr << cublasGetStatusString(status) << " " << func << '\n';
  }
}

template<typename Kernel>
void test_kernel(const char *kernel_name,
                 Kernel kernel,
                 int M, int N, int K, float alpha, float beta, int num_runs = 10) {
  auto allocator = mlkl::TensorAllocator();

  std::vector<int> s1{M, K};
  std::vector<int> s2{K, N};
  std::vector<int> s3{M, N};

  auto a_d = allocator.randn(s1, mlkl::Device::CUDA);
  auto b_d = allocator.randn(s2, mlkl::Device::CUDA);
  auto c_d = allocator.empty(s3, mlkl::Device::CUDA);

  auto a_cpu = allocator.empty(s1, mlkl::Device::CPU);
  auto b_cpu = allocator.empty(s2, mlkl::Device::CPU);
  auto c_cpu = allocator.empty(s3, mlkl::Device::CPU);
  auto ref_matrix = allocator.empty(s3, mlkl::Device::CPU);

  mlkl::copy(a_d, a_cpu);
  mlkl::copy(b_d, b_cpu);

  mlkl::sgemm(a_cpu, b_cpu, c_cpu, alpha, beta, mlkl::Device::CPU);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float total_duration = 0;

  // warm-up
  for (int i = 0; i < 10; ++i) {
    kernel(a_d, b_d, c_d, alpha, beta);
    CHECK_CUDA_ERROR();
  }

  for (int i = 0; i < num_runs; ++i) {
    mlkl::fill(c_d, 0);
    CHECK_CUDA_ERROR();

    hipEventRecord(start);

    kernel(a_d, b_d, c_d, alpha, beta);
    CHECK_CUDA_ERROR();

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_elapsed;
    hipEventElapsedTime(&time_elapsed, start, stop);
    total_duration += time_elapsed;
  }

  mlkl::to(c_d, mlkl::Device::CPU);

  CHECK_CUDA_ERROR();

  if (!mlkl::equals(c_d, ref_matrix)) {
    std::cerr << "Kernel " << kernel_name << " produced incorrect results." << std::endl;
  }

  float average_duration = total_duration / num_runs;
  float gflops = (2.0f * M * N * K) / (average_duration / 1000.0f) / 1e9;

  std::cout << "Kernel: " << kernel_name << " | "
            << "Size: " << M << "x" << K << "x" << N << " | "
            << "Time: " << average_duration << " ms | "
            << "GFLOPS: " << gflops << std::endl;

  // Cleanup
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void sgemm_cuda(int M, int N, int K, float alpha, float beta) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  int num_runs = 1000;

  auto cublas_kernel = [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) {
    int M = c.shape[0];
    int N = c.shape[1];
    int K = a.shape[1];
    CHECK_CUBLAS_STATUS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b.data, N, a.data, K, &beta, c.data, N));
  };
  // Test CUBLAS
  test_kernel("CUBLAS", cublas_kernel, M, N, K, alpha, beta, num_runs);

  // Test custom kernels
  test_kernel("SGEMM Kernel V2", [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) { mlkl::operators::cuda::sgemm_v2(a, b, c, alpha, beta); }, M, N, K, alpha, beta, num_runs);
  test_kernel("SGEMM Kernel V3", [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) { mlkl::operators::cuda::sgemm_v3(a, b, c, alpha, beta); }, M, N, K, alpha, beta, num_runs);
  test_kernel("SGEMM Kernel V4", [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) { mlkl::operators::cuda::sgemm_v4(a, b, c, alpha, beta); }, M, N, K, alpha, beta, num_runs);
  test_kernel("SGEMM Kernel V5", [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) { mlkl::operators::cuda::sgemm_v5(a, b, c, alpha, beta); }, M, N, K, alpha, beta, num_runs);
  test_kernel("SGEMM Kernel V6", [&](mlkl::Tensor &a, mlkl::Tensor &b, mlkl::Tensor &c, float alpha, float beta) { mlkl::operators::cuda::sgemm_v6(a, b, c, alpha, beta); }, M, N, K, alpha, beta, num_runs);

  hipblasDestroy(handle);
}